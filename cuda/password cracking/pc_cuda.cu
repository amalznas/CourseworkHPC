#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <time.h>
/****************************************************************************

  Compile with:
    nvcc -o pc_cuda pc_cuda.cu 
  Run with:
    ./pc_cuda

*****************************************************************************/

__device__ int is_a_match(char *attempt) {
	char plain_password1[] = "CV7828";
	char plain_password2[] = "FR8356";
	char plain_password3[] = "GS0484";
	char plain_password4[] = "TB4535";


	char *m = attempt;
	char *n = attempt;
	char *o = attempt;
	char *p = attempt;
	char *p1 = plain_password1;
	char *p2 = plain_password2;
	char *p3 = plain_password3;
	char *p4 = plain_password4;

	while(*m == *p1) { 
		if(*m == '\0') 
		{
			printf("Password: %s\n",plain_password1);
			break;
		}

		m++;
		p1++;
	}
	
	while(*n == *p2) { 
		if(*n == '\0') 
		{
			printf("Password: %s\n",plain_password2);
			break;
		}

		n++;
		p2++;
	}

	while(*o == *p3) { 
		if(*o == '\0') 
		{
			printf("Password: %s\n",plain_password3);
			break;
		}

		o++;
		p3++;
	}

	while(*p == *p4) { 
		if(*p == '\0') 
		{
			printf("Password: %s\n",plain_password4);
			return 1;
		}

		p++;
		p4++;
	}
	return 0;

}

__global__ void  kernel() {
	char i1,i2,i3,i4;

	char password[7];
	password[6] = '\0';

	int i = blockIdx.x+65;
	int j = threadIdx.x+65;
	char initialMatch = i; 
	char finalMatch = j; 

	password[0] = initialMatch;
	password[1] = finalMatch;
	for(i1='0'; i1<='9'; i1++){
		for(i2='0'; i2<='9'; i2++){
			for(i3='0'; i3<='9'; i3++){
				for(i4='0'; i4<='9'; i4++){
					password[2] = i1;
					password[3] = i2;
					password[4] = i3;
					password[5] = i4; 
					if(is_a_match(password)) {
					} 
					else {
	     			//printf("tried: %s\n", password);		  
					}
				}
			}
		}
	}
}

int time_difference(struct timespec *start, 
	struct timespec *finish, 
	long long int *difference) {
	long long int ds =  finish->tv_sec - start->tv_sec; 
	long long int dn =  finish->tv_nsec - start->tv_nsec; 
	if(dn < 0 ) {
		ds--;
		dn += 1000000000; 
	} 
	*difference = ds * 1000000000 + dn;
	return !(*difference > 0);
}

int main() {

	struct  timespec start, finish;
	long long int time_elapsed;
	clock_gettime(CLOCK_MONOTONIC, &start);

	kernel <<<26,26>>>();
	hipDeviceSynchronize();

	clock_gettime(CLOCK_MONOTONIC, &finish);
	time_difference(&start, &finish, &time_elapsed);
	printf("Time elapsed was %lldns or %0.9lfs\n", time_elapsed, (time_elapsed/1.0e9)); 

	return 0;
}


